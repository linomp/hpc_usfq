#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""

//The number of character in the encrypted text
#define N 1024

void checkCUDAError(const char*);
int get_text_length(const char * fname);
void read_file(const char*, int*);
void write_file(const char*, int*, int );

#define A 15
#define B 27
#define M 128
#define A_MMI_M 111

__device__ int modulo(int a, int b){
	int r = a % b;
	r = (r < 0) ? r + b : r;
	return r;
}

__global__ void decrypt_kernel(int *d_input, int *d_output, int length)
{
	for (int i = 0; i< length; i++){
		char x = d_input[i];
		d_output[i] = 111*(x-B) % M; 
	}
}

__global__ void decrypt_multiblock_kernel(int *d_input, int *d_output, int length)
{
	// ...
}


int main(int argc, char *argv[])
{
	int *h_input, *h_output;
	int *d_input, *d_output;
	unsigned int size;

	const char * fname;

	if (argc < 2) printf("Debe ingresar el nombre del archivo\n");
	else
		fname = argv[argc-1];

	int length = get_text_length(fname);

	size = length * sizeof(int);

	// reservo memoria para h_input y h_output
	h_input = (int *)malloc(size);
	h_output = (int *)malloc(size);

	// reservar memoria en la GPU para d_input y d_output
	hipMalloc(&d_input, size); hipMalloc(&d_output, size);
	checkCUDAError("Memory allocation");

	// leo el archivo con el mensaje cifrado
	read_file(fname, h_input);

	// transferir el arreglo de entrada al dispositivo
	hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
	checkCUDAError("Input transfer to device");

	// configurar la grilla de threads
	dim3 blocksPerGrid (1, 1, 1) 
	dim3 threadsPerBlock (N, 1, 1)

	// ejecutar el kernel
	decrypt_kernel <<< blocksPerGrid, threadsPerBlock >>>( d_input, d_output, length );

	hipDeviceSynchronize();
	checkCUDAError("Kernel execution");


	// transferir el contenido de d_output a la memoria de la CPU
	hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);
	checkCUDAError("Result transfer to host");

	printf("Texto desencriptado:\n");

	for (int i = 0; i < length; i++) {
		printf("%c", (char)h_output[i]); 
	}
	printf("\n");

	// liberar memoria en el dispositivo para d_input y d_output
	hipFree(d_input); hipFree(d_output);
	checkCUDAError("Free memory");

	/* free host buffers */
	free(h_input);
	free(h_output);

	return 0;
}


void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}


	
int get_text_length(const char * fname)
{
	FILE *f = NULL;
	f = fopen(fname, "r"); 

	size_t pos = ftell(f);    
	fseek(f, 0, SEEK_END);    
	size_t length = ftell(f); 
	fseek(f, pos, SEEK_SET);  

	fclose(f);

	return length;
}

void read_file(const char * fname, int* input)
{
	// printf("leyendo archivo %s\n", fname );

	FILE *f = NULL;
	f = fopen(fname, "r"); 
	if (f == NULL){
		fprintf(stderr, "Error: Could not find %s file \n", fname);
		exit(1);
	}

	int c; 
	while ((c = getc(f)) != EOF) {
		*(input++) = c;
	}

	fclose(f);
}

void write_file(const char * fname, int* input, int length)
{
	FILE *f = NULL;
	f = fopen(fname, "w"); 
	if (f == NULL){
		fprintf(stderr, "Error: Could not find %s file \n", fname);
		exit(1);
	}

	for (int i = 0; i < length; ++i)
	{
		putc((char)input[i],f);
	}

	fclose(f);
}