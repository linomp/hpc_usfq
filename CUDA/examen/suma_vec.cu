#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""

//The number of character in the encrypted text
#define N 1024

void checkCUDAError(const char*);
int get_text_length(const char * fname); 
 
__global__ void add_elements_kernel(float *d_input_1, float *d_input_2, float *d_output, int length)
{
	// qué thread soy?
	// aplicando offset porque hay más bloques
  //int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int idx = threadIdx.x;
	// verificar, xq pueden lanzarse más hilos que elementos
	if( idx < length ){ 
		float res = d_input_1[idx] + d_input_2[idx];
		// calcular y guardar valor en indice correspondiente en el buffer de salida
		d_output[idx] = res; 
	}
}


int main(int argc, char *argv[])
{
  float v1[5] = {1000.0, 2.0, 3.4, 7.0, 50.0};
  float v2[5] = {1.0, 4.0, 4, 1, 50.0};
  float *h_output;
	float *d_input_1, *d_input_2, *d_output;
	unsigned int size; 
  
	float t_deviceToHost, t_kernel, t_hostToDevice;
  struct timeval t_i, t_dth, t_k, t_htd; 

  int length = sizeof(v1)/sizeof(float); 

	size = length * sizeof(float);

	// reservo memoria para h_input y h_output
	//v1 = (float *)malloc(size);
  //v2 = (float *)malloc(size);
  h_output = (float *)malloc(size);

	// reservar memoria en la GPU para d_input y d_output
  hipMalloc(&d_input_1, size); 
  hipMalloc(&d_input_2, size); 
  hipMalloc(&d_output, size); 

  // transferir el arreglo de entrada al dispositivo
  gettimeofday(&t_i, NULL);
  hipMemcpy(d_input_1, v1, size, hipMemcpyHostToDevice);   
  hipMemcpy(d_input_2, v2, size, hipMemcpyHostToDevice); 
  gettimeofday(&t_htd, NULL);

  t_hostToDevice = (double)t_htd.tv_sec * 1000.0 + (double)t_htd.tv_usec / 1000.0 -
           ((double)t_i.tv_sec * 1000.0 + (double)t_i.tv_usec / 1000.0);

  // configurar la grilla de threads
  //dim3 blocksPerGrid ( (int) ceil(length/N), 1, 1) ;
  dim3 blocksPerGrid (1, 1, 1) ;
	dim3 threadsPerBlock (N, 1, 1);

  // ejecutar el kernel
  gettimeofday(&t_i, NULL);
	add_elements_kernel <<< blocksPerGrid, threadsPerBlock >>>( d_input_1, d_input_2, d_output, length );

	// sólo para medir tiempos, porque el memcopy ya sincroniza internamente
  hipDeviceSynchronize(); 
  gettimeofday(&t_k, NULL);
  t_kernel = (double)t_k.tv_sec * 1000.0 + (double)t_k.tv_usec / 1000.0 -
           ((double)t_i.tv_sec * 1000.0 + (double)t_i.tv_usec / 1000.0);

	// transferir el contenido de d_output a la memoria de la CPU
  gettimeofday(&t_i, NULL);
  hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost); 
  gettimeofday(&t_dth, NULL);
  t_deviceToHost = (double)t_dth.tv_sec * 1000.0 + (double)t_dth.tv_usec / 1000.0 -
           ((double)t_i.tv_sec * 1000.0 + (double)t_i.tv_usec / 1000.0);

	printf("Suma (GPU):\n");
	for (int i = 0; i < length; i++) {
		printf("%.2f, ", (float)h_output[i]); 
	}
  printf("\n");
  
  printf("Suma (Host):\n");
  verifyInHost(v1, v2, length);

  printf("\nTiempo transf. host-to-device %f ms\nTiempo kernel %f ms\nTiempo transf. device-to-host % f ms\n ", t_hostToDevice, t_kernel, t_deviceToHost);


	// liberar memoria en el dispositivo para d_input y d_output
  hipFree(d_input_1); 
  hipFree(d_input_2);
  hipFree(d_output); 
  
	/* free host buffers */
	//free(v1);
	//free(v2);

	return 0;
} 

void verifyInHost(float* v1, float* v2, int length){
  for (int i = 0; i < length; i++) {
		printf("%.2f, ", v1[i] + v2[i]); 
  }
  printf("\n");
}