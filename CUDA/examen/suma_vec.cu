#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""

//The number of character in the encrypted text
#define N 1024

void checkCUDAError(const char*);
int get_text_length(const char * fname); 
 
__global__ void add_elements_kernel(float *d_input_1, float *d_input_2, float *d_output, int length)
{
	// qué thread soy?
	// aplicando offset porque hay más bloques
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	// verificar, xq pueden lanzarse más hilos que elementos
	if( idx < length ){ 
		float res = d_input_1[idx] + d_input_2[idx];
		// calcular y guardar valor en indice correspondiente en el buffer de salida
		d_output[idx] = res; 
	}
}


int main(int argc, char *argv[])
{
  float v1[5] = {1000.0, 2.0, 3.4, 7.0, 50.0};
  float v2[5] = {1.0, 4.0, 4, 1, 50.0};
  float *h_output;
	float *d_input_1, *d_input_2, *d_output;
	unsigned int size; 

	int length = sizeof(v1)/sizeof(v1[0]);
	size = length * sizeof(float);

	// reservo memoria para h_input y h_output
	//v1 = (float *)malloc(size);
  //v2 = (float *)malloc(size);
  h_output = (float *)malloc(size);

	// reservar memoria en la GPU para d_input y d_output
  hipMalloc(&d_input_1, size); 
  hipMalloc(&d_input_2, size); 
  hipMalloc(&d_output, size); 

	// transferir el arreglo de entrada al dispositivo
  hipMemcpy(d_input_1, v1, size, hipMemcpyHostToDevice);   
	hipMemcpy(d_input_2, v2, size, hipMemcpyHostToDevice); 

  // configurar la grilla de threads
	dim3 blocksPerGrid ( (int) ceil(length/N), 1, 1) ;
	dim3 threadsPerBlock (N, 1, 1);

	// ejecutar el kernel
	add_elements_kernel <<< blocksPerGrid, threadsPerBlock >>>( d_input_1, d_input_2, d_output, length );

	// sólo para medir tiempos, porque el memcoy ya sincroniza internamente
	hipDeviceSynchronize(); 

	// transferir el contenido de d_output a la memoria de la CPU
	hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost); 

	printf("Texto desencriptado:\n");

	for (int i = 0; i < length; i++) {
		printf("%.2f", (float)h_output[i]); 
	}
	printf("\n");

	// liberar memoria en el dispositivo para d_input y d_output
  hipFree(d_input_1); 
  hipFree(d_input_2);
  hipFree(d_output); 
  
	/* free host buffers */
	//free(v1);
	//free(v2);

	return 0;
} 